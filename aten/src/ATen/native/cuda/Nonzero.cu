#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <ATen/cuda/EmptyTensor.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh> //for MAX_DIMS
#include <ATen/cuda/cub.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty_native.h>
#include <ATen/ops/nonzero_native.h>
#endif


namespace at::native {

namespace {
// Wrapper for DeviceSelect::If to handle tensors larger than INT_MAX
// Imported from https://github.com/NVIDIA/cccl/pull/1379
// #TODO: Remove the wrapper when https://github.com/NVIDIA/cccl/issues/1422 is released
template <typename InputIteratorT,
          typename OutputIteratorT,
          typename NumSelectedIteratorT,
          typename OffsetT,
          typename SelectOp>
static hipError_t dispatch_select_if_wrapper(
  void* d_temp_storage,
  std::size_t& temp_storage_bytes,
  InputIteratorT d_in,
  OutputIteratorT d_out,
  NumSelectedIteratorT d_num_selected_out,
  OffsetT num_items,
  SelectOp select_op,
  hipStream_t stream = 0)
{
    using flag_iterator_t = hipcub::NullType*;
    using equality_op_t   = hipcub::NullType;

    return hipcub::DispatchSelectIf<
        InputIteratorT,
        flag_iterator_t,
        OutputIteratorT,
        NumSelectedIteratorT,
        SelectOp,
        equality_op_t,
        OffsetT,
        false>::Dispatch(d_temp_storage,
                         temp_storage_bytes,
                         d_in,
                         nullptr,
                         d_out,
                         d_num_selected_out,
                         select_op,
                         equality_op_t{},
                         num_items,
                         stream);
}

template<typename T>
struct NonZeroOp
{
    __host__ __device__ __forceinline__ bool operator()(const T& a) const {
        return (a != T(0));
    }
};

template<typename T>
struct NonZeroOp<c10::complex<T>>
{
    __host__ __device__ __forceinline__ bool operator()(const c10::complex<T>& a) const {
        return (a.real() != T(0) || a.imag() != T(0));
    }
};

//TODO: actually support int64_t index_t
template<typename index_t>
struct TensorDims {
    index_t sizes[MAX_DIMS];
};

template <typename index_t>
__global__ void write_indices(
    int64_t* inp,
    TensorDims<index_t> dims,
    int ndim,
    index_t n) {
  auto index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    index_t div = 1;
    int64_t idx_flat = inp[index];
    #pragma unroll
    for (int dim = MAX_DIMS; dim >= 0; dim--) {
      if (dim > ndim - 1)
        continue;
      auto dim_size = dims.sizes[dim];
      inp[index + dim * n] = (idx_flat / div) % dim_size;
      div *= dim_size;
    }
  }
}

} //anonymous namespace

template<typename scalar_t>
void nonzero_cuda_out_impl(const Tensor& self, Tensor& out) {
  Tensor self_ = self.contiguous();
  int64_t N = self_.numel();  // Changed to int64_t to handle larger sizes
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  //Compute number of nonzero elements
  size_t temp_storage_bytes = 0;
  auto& allocator = *c10::cuda::CUDACachingAllocator::get();
  auto num_nonzeros = allocator.allocate(sizeof(int));
  hipcub::TransformInputIterator<bool, NonZeroOp<scalar_t>, const scalar_t*> itr(self_.const_data_ptr<scalar_t>(), NonZeroOp<scalar_t>());
  hipcub::DeviceReduce::Sum(nullptr, temp_storage_bytes, itr, (int*)num_nonzeros.get(), N, stream);
  auto temp_storage = allocator.allocate(temp_storage_bytes);
  hipcub::DeviceReduce::Sum(temp_storage.get(), temp_storage_bytes, itr, (int*)num_nonzeros.get(), N, stream);
  int num_nonzeros_h;
  at::cuda::memcpy_and_sync(&num_nonzeros_h, num_nonzeros.get(), sizeof(int), hipMemcpyDeviceToHost, stream);
  //expected output size is num_nonzeros x ndim
  //we are producing output with size {num_nonzeros, ndim} and strides {1, num_nonzeros} (that is, transposed ndim x num_nonzeros output)
  //we are able to directly use passed output with this size and strides, and we can also (per contract)
  //resize passed output with incorrect sizes anyway we want.
  //However, out with correct sizes and incorrect strides will have to be copied to from the intermediate we've produced.
  bool need_to_copy = out.dim() == 2 && out.sizes()[0] == num_nonzeros_h && out.sizes()[1] == self.dim() && !out.t().is_contiguous();
  at::Tensor out_temp = need_to_copy ?
      Tensor(at::detail::empty_cuda({self.dim(), num_nonzeros_h}, out.options())) :
      out.resize_({self.dim(), num_nonzeros_h});
  //Scalars are expected to produce output of size (1,0), so we can't write to it
  if (self.dim() > 0) {
    hipcub::CountingInputIterator<int64_t> counting_itr(0);
    temp_storage_bytes = 0;
    dispatch_select_if_wrapper(nullptr, temp_storage_bytes, counting_itr, out_temp.mutable_data_ptr<int64_t>(),
        (int*)num_nonzeros.get(), N, NonZeroOp<scalar_t>(), stream);
    temp_storage = allocator.allocate(temp_storage_bytes);
    dispatch_select_if_wrapper(temp_storage.get(), temp_storage_bytes, counting_itr, out_temp.mutable_data_ptr<int64_t>(),
        (int*)num_nonzeros.get(), N, NonZeroOp<scalar_t>(), stream);
    if (num_nonzeros_h > 0 && self.dim() > 1) {
      TensorDims<int> dims;
      for (int i = 0; i < self.dim(); i++) {
          dims.sizes[i] = self.sizes()[i];
      }
      const int nthreads = 256;
      const int nblocks = (num_nonzeros_h + nthreads - 1) / nthreads;
      write_indices<<<nblocks, nthreads, 0, stream>>>(out_temp.mutable_data_ptr<int64_t>(),
      dims, self.dim(), num_nonzeros_h);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
  if (need_to_copy) {
    out.copy_(out_temp.t());
  } else {
    //transpose out so it is correct size
    Tensor out_ = out_temp.t();
    out.set_(out_);
  }
}

Tensor& nonzero_out_cuda(const Tensor& self, Tensor& out) {
  TORCH_CHECK(self.numel() < std::numeric_limits<int64_t>::max(), "nonzero is not supported for tensors with more than INT_MAX elements, \
  See https://github.com/pytorch/pytorch/issues/51871");
  TORCH_CHECK(out.dtype() == at::kLong, "Expected object of scalar type ", at::kLong, " as out, but got ", out.dtype());
  TORCH_CHECK(self.device() == out.device(), "expected self and out to be on the same device, but got out on ",
  out.device(), " and self on ", self.device());
  TORCH_CHECK(self.dim() <= MAX_DIMS, "nonzero is not supported for tensor with more than ", MAX_DIMS, " dimensions");
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND4(at::ScalarType::ComplexHalf, at::ScalarType::Bool, at::ScalarType::BFloat16, at::ScalarType::Half,
      self.scalar_type(), "nonzero_cuda",
      [&] {nonzero_cuda_out_impl<scalar_t>(self, out);});
  return out;
}

Tensor nonzero_cuda(const Tensor& self) {
  Tensor out = at::detail::empty_cuda({0}, self.options().dtype(kLong));
  return at::native::nonzero_out_cuda(self, out);
}
} //namespace at::native
